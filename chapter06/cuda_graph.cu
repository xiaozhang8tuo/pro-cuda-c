#include "hip/hip_runtime.h"
// nvcc -o graph cuda_graph.cu -lcudart -std=c++11
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void debug_print(int *a) {
    if (a != NULL) {
        printf("in kernel func, a = %d\n", *a);
    } else {
        printf("in kernel func, received NULL pointer\n");
    }
}

int main() {
    int a = 0;
    int *d_a = NULL;
    hipError_t cudaStatus;

    // CUDA 流和图
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t instance;

    // 创建 CUDA 流
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    // 执行 CPU 指令
    a = 1;
    printf("1 graph a: %d\n", a);

    // 开始捕获图
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipStreamDestroy(stream);
        return 1;
    }
    a = 3;
    printf("2 graph a: %d\n", a);


    // 将变量a拷贝到设备内存
    cudaStatus = hipMallocAsync((void**)&d_a, sizeof(int), stream);
    // cudaStatus = hipMalloc((void**)&d_a, sizeof(int));//hipMalloc failed: operation not permitted when stream is capturing
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        hipStreamDestroy(stream);
        return 1;
    }

    // 将变量a拷贝到设备内存
    cudaStatus = hipMemcpyAsync(d_a, &a, sizeof(int), hipMemcpyHostToDevice, stream);
    // cudaStatus = hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice); // hipMemcpy failed: operation would make the legacy stream depend on a capturing blocking stream
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipStreamDestroy(stream);
        return 1;
    }

    // 在捕获图期间不要使用 hipMemcpy，将其移到捕获图之后
    debug_print<<<1, 1, 0, stream>>>(d_a);

    // 结束捕获图
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipStreamDestroy(stream);
        return 1;
    }

    a = 2;
    printf("3 graph a: %d\n", a);

    // 实例化并执行图
    cudaStatus = hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipStreamDestroy(stream);
        return 1;
    }
    cudaStatus = hipGraphLaunch(instance, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed: %s\n", hipGetErrorString(cudaStatus));
        hipGraphExecDestroy(instance);
        hipFree(d_a);
        hipStreamDestroy(stream);
        return 1;
    }
    hipStreamSynchronize(stream);
    printf("4 graph a: %d\n", a);

    // 清理资源
    hipGraphExecDestroy(instance);
    hipFree(d_a);
    hipStreamDestroy(stream);


    return 0;
}
